#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <complex>
#include <assert.h>
#include <algorithm>
#include <hip/hip_complex.h>
#include "util_type.h"
#include "util_type_internal.h"
#include "util.cuh"
#include "memory_ops.h"
#include "memory_ops_device_functions.h"
#include "stat_ops.h"
#include "update_ops_cuda.h"

__host__ void* allocate_cuda_stream_host(unsigned int max_cuda_stream) {
	hipStream_t* stream = (hipStream_t*)malloc(max_cuda_stream * sizeof(hipStream_t));
	for (unsigned int i = 0; i < max_cuda_stream; ++i) hipStreamCreate(&stream[i]);
	void* hip_stream = reinterpret_cast<void*>(stream);
	return hip_stream;
}

__host__ void release_cuda_stream_host(void* hip_stream, unsigned int max_cuda_stream) {
	hipStream_t* stream = reinterpret_cast<hipStream_t*>(hip_stream);
	for (unsigned int i = 0; i < max_cuda_stream; ++i) hipStreamDestroy(stream[i]);
	free(stream);
}

__global__ void init_qstate(GTYPE* state_gpu, ITYPE dim){
	ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < dim) {
		state_gpu[idx] = make_hipDoubleComplex(0.0, 0.0);
	}
	if (idx == 0) state_gpu[idx] = make_hipDoubleComplex(1.0, 0.0);
}

// void* (GTYPE*)
__host__ void* allocate_quantum_state_host(ITYPE dim){
	GTYPE *state_gpu;
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipMalloc((void**)&state_gpu, dim * sizeof(GTYPE)));
	void* psi_gpu = reinterpret_cast<void*>(state_gpu);
    return psi_gpu;
}

__host__ void initialize_quantum_state_host(void* state, ITYPE dim, void* stream) {
	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	hipError_t cudaStatus;
	unsigned int block = dim <= 1024 ? dim : 1024;
	unsigned int grid = dim / block;
	init_qstate << <grid, block, 0, *hip_stream >> > (state_gpu, dim);

	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	cudaStatus = hipGetLastError();
	checkCudaErrors(cudaStatus, __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	stream = reinterpret_cast<void*>(hip_stream);
}

__host__ void initialize_quantum_state_host(void* state, ITYPE dim) {
	hipStream_t hip_stream = (hipStream_t)0;
	initialize_quantum_state_host(state, dim, &hip_stream);
}

__host__ void release_quantum_state_host(void* state){
	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	checkCudaErrors(hipFree(state_gpu), __FILE__, __LINE__);
}

__host__ void initialize_Haar_random_state_host(void *state, ITYPE dim, void* stream) {
	initialize_Haar_random_state_with_seed_host(state, dim, (unsigned)time(NULL), stream);
}

__host__ void initialize_Haar_random_state_host(void *state, ITYPE dim) {
	initialize_Haar_random_state_with_seed_host(state, dim, (unsigned)time(NULL));
}

__global__ void init_rnd(hiprandState *const rnd_state, const unsigned int seed)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, tid, 0, &rnd_state[tid]);
}

/*
__global__ void rand_normal_mtgp32(hiprandState* rnd_state, GTYPE* state, ITYPE dim){
	ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
    double2 rnd;
    hiprandStateMtgp32 localState = rnd_state[idx];
	if (idx < dim) {
        rnd = hiprand_normal2_double(&localState);
        state[idx] = make_hipDoubleComplex(rnd.x, rnd.y);
        rnd_state[idx] = localState;
    }
}
*/

__global__ void rand_normal_xorwow(hiprandState* rnd_state, GTYPE* state, ITYPE dim){
	ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
    // double2 rnd;
    double tmp1, tmp2;
    double real, imag;
    hiprandStateXORWOW localState = rnd_state[idx];
	if (idx < dim) {
        // rnd = hiprand_normal2_double(&localState);
        tmp1 = hiprand_uniform_double(&localState);
        tmp2 = hiprand_uniform_double(&localState);
	    real = sqrt(-1.0*log(tmp1)) * sinpi(2.0*tmp2);
        tmp1 = hiprand_uniform_double(&localState);
        tmp2 = hiprand_uniform_double(&localState);
	    imag = sqrt(-1.0*log(tmp1)) * sinpi(2.0*tmp2);
        state[idx] = make_hipDoubleComplex(real, imag);
        rnd_state[idx] = localState;
    }
}

__host__ void initialize_Haar_random_state_with_seed_host(void *state, ITYPE dim, UINT seed, void* stream) {
	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	//const ITYPE ignore_first = 40;
	double norm = 0.;

	hiprandState* rnd_state;
	checkCudaErrors(hipMalloc((void**)&rnd_state, dim * sizeof(hiprandState)), __FILE__, __LINE__);

	// HIPRAND_RNG_PSEUDO_XORWOW
	// HIPRAND_RNG_PSEUDO_MT19937 offset cannot be used and need sm_35 or higher.

	unsigned int block = dim <= 512 ? dim : 512;
	unsigned int grid = min((int)(dim / block), 512);

	init_rnd << < grid, block, 0, *hip_stream >> > (rnd_state, seed);
	checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);

	rand_normal_xorwow << < grid, block, 0, *hip_stream >> > (rnd_state, state_gpu, dim);
	checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);

	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipFree(rnd_state), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);

	norm = state_norm_host(state, dim, hip_stream);
	normalize_host(norm, state, dim, hip_stream);
}

__host__ void initialize_Haar_random_state_with_seed_host(void *state, ITYPE dim, UINT seed) {
	hipStream_t hip_stream = (hipStream_t)0;
	initialize_Haar_random_state_with_seed_host(state, dim, seed, &hip_stream);
}
